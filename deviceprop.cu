#include <iostream>
#define cout std::cout
#define endl std::endl;
int main(void)
{
	hipDeviceProp_t prop;
	int count;
	cudaErrorCheck(hipGetDeviceCount(&count));
	for(int i=0;i<count;i++)
	{
		cout<<"Printing details about device "<<i<<endl;
		cudaErrorCheck(cudaGetDeviceProp(&prop,i));
		cout<<"Name: "<<prop.name<<endl;
		cout<<"Total Global Memory: "<<prop.totalGlobalMem<<endl;
		cout<<"Registers per block: "<<prop.regsPerBlock<<endl;
		cout<<"Warp Size: "<<prop.warpSize<<endl;
		cout<<"Max Threads Per Block: "<<prop.maxThreadsPerBlock<<endl;
		cout<<"Max Thread Dimension: "<<prop.maxThreadsDim[0]<<", "<<prop.maxThreadsDim[1]<<", "<<prop.maxThreadsDim[2]<<", "<<endl;
		cout<<"Max Grid Size: "<<prop.maxGridSize[0]<<", "<<prop.maxGridSize[1]<<", "<<prop.maxGridSize[2]<<", "<<endl;
		cout<<"Multi Processor Count: "<<prop.multiProcessorCount<<endl;
	}
}